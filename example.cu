#include <iostream>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


using namespace std;

struct Address
{
    int numa;
    int numb;
};

__global__ void  pairhmm( Address * address, int * result_d)
{

    clock_t start_time=clock();
    int warp_index=threadIdx.x/32;
    int numa=address[warp_index]. numa;
    int numb=address[warp_index]. numb;
    int result=0;
 
    int round=0;
   for(round=0;round<2;round++)
  {   
    for(int i=0;i<numa;i++)
    {
 	if(threadIdx.x%32==0) printf("round=%d   warp %d  numa=%d  i=%d \n",round,  warp_index, numa,i);       
        for(int j=0;j<numb;j++)
        {
        
 	if(threadIdx.x%32==0) printf("warp %d            numb=%d  j=%d \n", warp_index, numb,j);       
        
    	result+=i+j*2;    
        }
    }
  
    if(threadIdx.x%32==0) printf("round=%d  warp %d endendend \n",round,  warp_index);       
    result_d[threadIdx.x]=result;
  }
    clock_t finish_time=clock();  
    int time=(int)( finish_time-start_time);	
    if(threadIdx.x%32==0)  	printf("%d\n", time);

}

int main()
{

    Address * address;
    address=(Address *)malloc(sizeof(Address)* 4);
    address[0].numa=2;
    address[0].numb=2;
  
    address[1].numa=4;
    address[1].numb=4;
    
    address[2].numa=6;
    address[2].numb=6;
    
    address[3].numa=8;
    address[3].numb=8;
    Address * address_d;

    hipMalloc( (Address **)&address_d,sizeof(int) *100 );
    hipMemcpy(address_d,address,4*sizeof(Address), hipMemcpyHostToDevice);
    
    int blocksize=64;
    int gridsize=1;
    
    int *result_h;
    int *result_d;
    
    result_h=(int *) malloc( sizeof(int)* 128);
    hipMalloc( (int **)&result_d,sizeof(int) *128);
    pairhmm<<<gridsize,blocksize>>>(address_d,result_d);
    hipMemcpy(result_h,result_d,128*sizeof(int), hipMemcpyDeviceToHost);
    
   // for(int i=0;i<128;i++)
   //   printf("index= %d %d\n", i, result_h[i]);
    
    hipDeviceSynchronize();


    return 0;
}
